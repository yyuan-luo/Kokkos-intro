
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>

#define M 12500
#define N 12500
#define BLOCK_SIZE 256 // Define the block size for CUDA kernel

__global__ void matrixMultiplication(double* matrix1, double* matrix2, double* result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        double sum = 0.0;
        for (int k = 0; k < n; ++k) {
            sum += matrix1[row * n + k] * matrix2[k + col * n];
        }
        result[row * n + col] = sum;
    }
}

int main() {
    double* matrix1 = new double[M * N];
    double* matrix2 = new double[N * M];
    double* result = new double[M * M];

    for (int i = 0; i < M * N; i++) {
        matrix1[i] = 1.0;
    }

    for (int i = 0; i < N * M; i++) {
        matrix2[i] = 2.0;
    }

    double* d_matrix1;
    double* d_matrix2;
    double* d_result;

    hipMalloc((void**)&d_matrix1, M * N * sizeof(double));
    hipMalloc((void**)&d_matrix2, N * M * sizeof(double));
    hipMalloc((void**)&d_result, M * M * sizeof(double));

    hipMemcpy(d_matrix1, matrix1, M * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, N * M * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    struct timeval start, end;
    gettimeofday(&start, nullptr);

    matrixMultiplication<<<gridDim, blockDim>>>(d_matrix1, d_matrix2, d_result, M);

    gettimeofday(&end, nullptr);
    double elapsedTime = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

    std::cout << "Time: " << elapsedTime << "s" << std::endl;

    hipMemcpy(result, d_result, M * M * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_result);

    delete[] matrix1;
    delete[] matrix2;
    delete[] result;

    return 0;
}
