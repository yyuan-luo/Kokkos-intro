
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <chrono>
#include <iostream>

#define BLOCK_SIZE 256  // Define the block size for CUDA kernel

__global__ void matrixMultiplication(double* matrix1, double* matrix2, double* result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        double sum = 0.0;
        for (int k = 0; k < n; ++k) {
            sum += matrix1[row * n + k] * matrix2[k + col * n];
        }
        result[row * n + col] = sum;
    }
}

int main(int argc, char const* argv[]) {
    if (argc < 2) {
        printf("./main.cuda (M=1250) (N=1250)\n");
    }
    unsigned long int M = 1250;
    unsigned long int N = 1250;
    if (argc == 3) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
    }
    unsigned long long flops_count = 2 * N * N * N;
    struct timeval start, end;
    gettimeofday(&start, nullptr);

    double* matrix1 = new double[M * N];
    double* matrix2 = new double[N * M];
    double* result = new double[M * M];

    for (int i = 0; i < M * N; i++) {
        matrix1[i] = 1.0;
    }

    for (int i = 0; i < N * M; i++) {
        matrix2[i] = 2.0;
    }

    double* d_matrix1;
    double* d_matrix2;
    double* d_result;

    hipMalloc((void**)&d_matrix1, M * N * sizeof(double));
    hipMalloc((void**)&d_matrix2, N * M * sizeof(double));
    hipMalloc((void**)&d_result, M * M * sizeof(double));

    hipMemcpy(d_matrix1, matrix1, M * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, N * M * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    auto kernel_start = std::chrono::high_resolution_clock::now();

    matrixMultiplication<<<gridDim, blockDim>>>(d_matrix1, d_matrix2, d_result, M);

    auto kernel_end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(kernel_end - kernel_start);

    double flops = flops_count / (duration.count() * 1e-6);
    std::cout << "Kernel: " << (duration.count() * 1e-6) << "s" << std::endl;
    std::cout << "FLOPS: " << flops << std::endl;

    hipMemcpy(result, d_result, M * M * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_result);

    delete[] matrix1;
    delete[] matrix2;
    delete[] result;

    gettimeofday(&end, nullptr);
    double elapsedTime = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

    std::cout << "Time: " << elapsedTime << "s" << std::endl;

    return 0;
}
