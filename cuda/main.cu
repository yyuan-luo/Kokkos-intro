
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <iostream>

#define BLOCK_SIZE 32  // Define the block size for CUDA kernel

__global__ void matrixMultiplication(double* matrix1, double* matrix2, double* result, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < K) {
        double sum = 0.0;
        for (int i = 0; i < N; ++i) {
            sum += matrix1[row * N + i] * matrix2[i * K + col];
        }
        result[row * K + col] = sum;
    }
}

int main(int argc, char const* argv[]) {
    if (argc < 4 && argc > 1) {
        printf("./main.cuda (M=1250) (N=1250) (K=1250)\n");
    }
    unsigned long int M = 1250;
    unsigned long int N = 1250;
    unsigned long int K = 1250;
    if (argc == 4) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
    }
    unsigned long long flops_count = 2 * M * N * K;
    struct timeval start, end;
    gettimeofday(&start, nullptr);

    double* matrix1 = new double[M * N];
    double* matrix2 = new double[N * K];
    double* result = new double[M * K];

    for (int i = 0; i < M * N; i++) {
        matrix1[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    for (int i = 0; i < N * K; i++) {
        matrix2[i] = static_cast<double>(rand()) / RAND_MAX;
    }

    double* d_matrix1;
    double* d_matrix2;
    double* d_result;

    hipMalloc((void**)&d_matrix1, M * N * sizeof(double));
    hipMalloc((void**)&d_matrix2, N * K * sizeof(double));
    hipMalloc((void**)&d_result, M * K * sizeof(double));

    hipMemcpy(d_matrix1, matrix1, M * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, N * K * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (K + blockDim.y - 1) / blockDim.y);

    auto kernel_start = std::chrono::high_resolution_clock::now();

    matrixMultiplication<<<gridDim, blockDim>>>(d_matrix1, d_matrix2, d_result, N, K);

    auto kernel_end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(kernel_end - kernel_start);

    double flops = flops_count / (duration.count() * 1e-6);
    std::cout << "Kernel: " << (duration.count() * 1e-6) << "s" << std::endl;
    std::cout << "FLOPS: " << flops << std::endl;

    hipMemcpy(result, d_result, M * K * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_result);

    delete[] matrix1;
    delete[] matrix2;
    delete[] result;

    gettimeofday(&end, nullptr);
    double elapsedTime = (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1000000.0;

    std::cout << "Time: " << elapsedTime << "s" << std::endl;

    return 0;
}
